#include "hip/hip_runtime.h"
#include <fast_mesh_operations.h>
#include <thrust/device_vector.h>
#include <>

using namespace thrust;

namespace ab {

	__global__ void kernel_calculate_normals_no_weight(Vertex* vertices, HalfEdge* half_edges, float3* normals, unsigned vertice_count) {
		int stride = blockDim.x;
		int offset = threadIdx.x;
		//printf("BLOCK %d launched by the host with stride %d\n", offset,stride);
		//calculate normal without weight
		for (int i = offset; i < vertice_count; i+=stride) {
			auto& vert = vertices[i];
			if (vert.he == -1) {
				continue;
			}
			int he = vert.he;
			float3 normal;
			normal.x = 0.f;
			normal.y = 0.f;
			normal.z = 0.f;
			do {
				HalfEdge& halfedge = half_edges[he];
				float3 a = vertices[halfedge.origin].position;
				float3 b = vertices[half_edges[halfedge.next].origin].position;
				normal = normal + cross3df(a, b);
				he = halfedge.next;
			} while (he != vert.he);
			normals[i] = normalized(normal);
		}
	}

	void calculate_normals_he_parallel_no_weight(HalfedgeMesh* mesh) {
		mesh->normals.resize(mesh->vertices.size());
		thrust::device_vector<HalfEdge> halfedges = mesh->half_edges;
		thrust::device_vector<Vertex> vertices = mesh->vertices;
		thrust::device_vector<float3> normals = mesh->normals;
		kernel_calculate_normals_no_weight<<<1,128>>>(vertices.data().get(), halfedges.data().get(), normals.data().get(), vertices.size());
		hipDeviceSynchronize();
		printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
		thrust::copy(normals.begin(), normals.end(), mesh->normals.begin());
	}
}



