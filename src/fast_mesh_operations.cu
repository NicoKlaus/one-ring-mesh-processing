#include "hip/hip_runtime.h"
#include <fast_mesh_operations.h>
#include <thrust/device_vector.h>
#include <>

using namespace thrust;

namespace ab {

	__global__ void kernel_calculate_normals_no_weight(float3 positions,int* faces,int* face_indices,int* face_sizes, float3* normals, int face_count) {
		int stride = blockDim.x;
		int offset = threadIdx.x;
		for (int i = offset; i < face_count; i += stride) {
			int base_index = faces[i];
			int face_size = face_sizes[i];
			
			float3 point_a = positions[face_vertices[base_index+(face_size-1)]];
			float3 point_b = positions[face_vertices[base_index]];
			float3 edge_vector_ab = point_b-point_a;
			float3 normal;
			normal.x = 0.f;
			normal.y = 0.f;
			normal.z = 0.f;
			//circulate trough the rest of the face and calculate the normal
			for (int j = 0;j< face_size;++j){
				float3 point_c = positions[face_vertices[base_index+((j+1)%face_size)]];
				float3 edge_vector_bc = point_c - point_b;
				//adding to the normal vector
				normal += cross3df(edge_vector_ab,edge_vector_bc);
				edge_vector_ab = edge_vector_bc;
			}
			//add to every vertice in the face
			for (int j = 0;j< face_size;++j){
				float3* vn = &normals[face_vertices[base_index+j]];
				atomicAdd_system(&vn->x, normal.x);
				atomicAdd_system(&vn->y, normal.y);
				atomicAdd_system(&vn->z, normal.z);
			}
		}
	}

	__global__ void kernel_calculate_normals_no_weight(Vertex* vertices, HalfEdge* half_edges, float3* normals, unsigned vertice_count) {
		int stride = blockDim.x;
		int offset = threadIdx.x;
		//printf("BLOCK %d launched by the host with stride %d\n", offset,stride);
		//calculate normal without weight
		for (int i = offset; i < vertice_count; i+=stride) {
			auto& vert = vertices[i];
			if (vert.he == -1) {
				continue;
			}
			int he = vert.he;
			float3 normal;
			normal.x = 0.f;
			normal.y = 0.f;
			normal.z = 0.f;
			do {
				HalfEdge& halfedge = half_edges[he];
				float3 a = vertices[halfedge.origin].position;
				float3 b = vertices[half_edges[halfedge.next].origin].position;
				normal = normal + cross3df(a, b);
				he = halfedge.next;
			} while (he != vert.he);
			normals[i] = normalized(normal);
		}
	}

	__global__ void kernel_calculate_face_centroids(Vertex* vertices, HalfEdge* half_edges,Loop* loops, float3* centroids, unsigned loop_count) {
		int stride = blockDim.x;
		int offset = threadIdx.x;
		for (int i = offset; i < loop_count; i += stride) {
			auto& loop = loops[i];
			if (loop.is_border) {
				continue;
			}
			int he = loop.he;
			float3 centroid;
			int edge_count = 0;
			centroid.x = 0.f;
			centroid.y = 0.f;
			centroid.z = 0.f;
			do {
				HalfEdge& halfedge = half_edges[he];
				float3 a = vertices[halfedge.origin].position;
				centroid = centroid + a;
				++edge_count;
				he = halfedge.next;
			} while (he != loop.he);
			centroid.x /= (float)edge_count;
			centroid.y /= (float)edge_count;
			centroid.z /= (float)edge_count;
			centroids[i] = centroid;
		}
	}

	void calculate_normals_he_parallel_no_weight(HalfedgeMesh* mesh) {
		mesh->normals.resize(mesh->vertices.size());
		thrust::device_vector<HalfEdge> halfedges = mesh->half_edges;
		thrust::device_vector<Vertex> vertices = mesh->vertices;
		thrust::device_vector<float3> normals = mesh->normals;
		kernel_calculate_normals_no_weight<<<1,128>>>(vertices.data().get(), halfedges.data().get(), normals.data().get(), vertices.size());
		hipDeviceSynchronize();
		printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
		thrust::copy(normals.begin(), normals.end(), mesh->normals.begin());
	}

	void calculate_centroids_he_parallel(HalfedgeMesh* mesh,std::vector<float3>& centroids_array) {
		centroids_array.resize(mesh->loops.size());
		thrust::device_vector<HalfEdge> halfedges = mesh->half_edges;
		thrust::device_vector<Vertex> vertices = mesh->vertices;
		thrust::device_vector<Loop> loops = mesh->loops;
		thrust::device_vector<float3> centroids = centroids_array;
		kernel_calculate_face_centroids <<<1, 128>>> (vertices.data().get(), halfedges.data().get(), loops.data().get(), centroids.data().get(), loops.size());
		hipDeviceSynchronize();
		printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
		thrust::copy(centroids.begin(), centroids.end(), centroids_array.begin());
	}
}



